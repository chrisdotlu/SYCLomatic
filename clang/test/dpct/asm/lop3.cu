// RUN: dpct -out-root %T/lop3 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/lop3/lop3.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/lop3/lop3.dp.cpp -o %T/lop3/lop3.dp.o %}
// clang-format off

#include <hip/hip_runtime.h>
#include <cstdint>

// a^b^c
static __device__ __forceinline__ uint32_t LOP3LUT_XOR(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t d1;
  // CHECK: d1 = a ^ b ^ c;
  asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(d1) : "r"(a), "r"(b), "r"(c));
  return d1;
}

// (a ^ (c & (b ^ a)))
static __device__ __forceinline__ uint32_t LOP3LUT_XORAND(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t d2;
  // CHECK: d2 = (a ^ (c & (b ^ a)));
  asm("lop3.b32 %0, %1, %3, %2, 0xb8;" : "=r"(d2) : "r"(a), "r"(b), "r"(c));
  return d2;
}

// ((a & (b | b)) | (b & b))
static __device__ __forceinline__ uint32_t LOP3LUT_ANDOR(uint32_t a, uint32_t b) {
  uint32_t d3;
  // CHECK: d3 = ((a & (b | b)) | (b & b));
  asm("lop3.b32 %0, %1, %2, %2, 0xe8;" : "=r"(d3) : "r"(a), "r"(b));
  return d3;
}

#define B 3
__device__  int hard(int a) {
  int d4;
  // CHECK: d4 = (~(a + B) & B & ~3) | (~(a + B) & B & 3) | ((a + B) & ~B & ~3);
  asm("lop3.b32 %0, %1, %2, 3, 0x1C;" : "=r"(d4) : "r"(a + B), "r"(B));
  return d4;
}
// clang-format on
