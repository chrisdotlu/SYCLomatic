// UNSUPPORTED: system-windows
// RUN: dpct --format-range=none -out-root %T/double2_overloaded_operator %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/double2_overloaded_operator/double2_overloaded_operator.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/double2_overloaded_operator/double2_overloaded_operator.dp.cpp -o %T/double2_overloaded_operator/double2_overloaded_operator.dp.o %}

#include <hip/hip_runtime.h>

// CHECK: typedef sycl::double2 ty;
typedef double2 ty;
// CHECK: typedef sycl::double2& ty2;
typedef double2& ty2;
// CHECK: typedef sycl::double2&& ty3;
typedef double2&& ty3;
// CHECK: typedef sycl::double2** ty4;
typedef double2** ty4;
// CHECK: typedef sycl::double2* ty5;
typedef double2* ty5;
// CHECK: typedef sycl::double2*** ty6;
typedef double2*** ty6;
// CHECK: typedef const sycl::double2*** ty7;
typedef const double2*** ty7;
// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 &operator*=(sycl::double2 &v, const sycl::double2 &v2)          ;
// CHECK: }  // namespace dpct_operator_overloading
inline double2 &operator*=(double2 &v, const double2 &v2)          ;

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 &operator*=(sycl::double2 &v, const sycl::double2 &v2)          ;
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
inline double2 &operator*=(double2 &v, const double2 &v2)          ;

// CHECK: inline sycl::double2 &copy(sycl::double2 &v, const sycl::double2 &v2) {
// CHECK:   v.x() = v2.x();
// CHECK:   v.y() = v2.y();
// CHECK:   return v;
// CHECK: }
__host__ __device__ inline double2 &copy(double2 &v, const double2 &v2) {
  v.x = v2.x;
  v.y = v2.y;
  return v;
}
// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 &operator+=(sycl::double2 &v, const sycl::double2 &v2) {
// CHECK:   v.x() += v2.x();
// CHECK:   v.y() += v2.y();
// CHECK:   return v;
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 &operator+=(double2 &v, const double2 &v2) {
  v.x += v2.x;
  v.y += v2.y;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 &operator+=(sycl::double2 &v, const sycl::double2 &v2) {
// CHECK:   v.x() += v2.x();
// CHECK:   v.y() += v2.y();
// CHECK:   return v;
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 &operator+=(double2 &v, const double2 &v2) {
  v.x += v2.x;
  v.y += v2.y;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 &operator-=(sycl::double2 &v, const sycl::double2 &v2) {
// CHECK:   v.x() -= v2.x();
// CHECK:   v.y() -= v2.y();
// CHECK:   return v;
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 &operator-=(double2 &v, const double2 &v2) {
  v.x -= v2.x;
  v.y -= v2.y;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 &operator-=(sycl::double2 &v, const sycl::double2 &v2) {
// CHECK:   v.x() -= v2.x();
// CHECK:   v.y() -= v2.y();
// CHECK:   return v;
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 &operator-=(double2 &v, const double2 &v2) {
  v.x -= v2.x;
  v.y -= v2.y;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 &operator*=(sycl::double2 &v, const double &r) {
// CHECK:   v.x() *= r;
// CHECK:   v.y() *= r;
// CHECK:   return v;
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 &operator*=(double2 &v, const double &r) {
  v.x *= r;
  v.y *= r;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 &operator*=(sycl::double2 &v, const double &r) {
// CHECK:   v.x() *= r;
// CHECK:   v.y() *= r;
// CHECK:   return v;
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 &operator*=(double2 &v, const double &r) {
  v.x *= r;
  v.y *= r;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 &operator/=(sycl::double2 &v, const double &r) {
// CHECK:   v.x() /= r;
// CHECK:   v.y() /= r;
// CHECK:   return v;
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 &operator/=(double2 &v, const double &r) {
  v.x /= r;
  v.y /= r;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 &operator/=(sycl::double2 &v, const double &r) {
// CHECK:   v.x() /= r;
// CHECK:   v.y() /= r;
// CHECK:   return v;
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 &operator/=(double2 &v, const double &r) {
  v.x /= r;
  v.y /= r;
  return v;
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline bool operator==(const sycl::double2 &v1,
// CHECK:                                            const sycl::double2 &v2) {
// CHECK:   return ((v1.x() == v2.x()) && (v1.y() == v2.y()));
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline bool operator==(const double2 &v1,
                                           const double2 &v2) {
  return ((v1.x == v2.x) && (v1.y == v2.y));
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline bool operator==(const sycl::double2 &v1,
// CHECK:                                            const sycl::double2 &v2) {
// CHECK:   return ((v1.x() == v2.x()) && (v1.y() == v2.y()));
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline bool operator==(const double2 &v1,
                                           const double2 &v2) {
  return ((v1.x == v2.x) && (v1.y == v2.y));
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline bool operator!=(const sycl::double2 &v1,
// CHECK:                                            const sycl::double2 &v2) {
// CHECK:   return (!(dpct_operator_overloading::operator==(v1 , v2)));
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline bool operator!=(const double2 &v1,
                                           const double2 &v2) {
  return (!(v1 == v2));
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline bool operator!=(const sycl::double2 &v1,
// CHECK:                                            const sycl::double2 &v2) {
// CHECK:   return (!(dpct_operator_overloading::operator==(v1 , v2)));
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline bool operator!=(const double2 &v1,
                                           const double2 &v2) {
  return (!(v1 == v2));
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: inline sycl::double2 operator+(const sycl::double2 &v) { return v; }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator+(const double2 &v) { return v; }

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator+(const sycl::double2 &v) { return v; }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator+(const double2 &v) { return v; }

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: inline sycl::double2 operator-(const sycl::double2 &v) {
// CHECK:   return sycl::double2(-v.x(), -v.y());
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator-(const double2 &v) {
  return make_double2(-v.x, -v.y);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator-(const sycl::double2 &v) {
// CHECK:   return sycl::double2(-v.x(), -v.y());
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator-(const double2 &v) {
  return make_double2(-v.x, -v.y);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 operator+(const sycl::double2 &v1,
// CHECK:                                              const sycl::double2 &v2) {
// CHECK:   return sycl::double2(v1.x() + v2.x(), v1.y() + v2.y());
// CHECK: }
// CHECK:}  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator+(const double2 &v1,
                                             const double2 &v2) {
  return make_double2(v1.x + v2.x, v1.y + v2.y);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator+(const sycl::double2 &v1,
// CHECK:                                              const sycl::double2 &v2) {
// CHECK:   return sycl::double2(v1.x() + v2.x(), v1.y() + v2.y());
// CHECK: }
// CHECK:}  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator+(const double2 &v1,
                                             const double2 &v2) {
  return make_double2(v1.x + v2.x, v1.y + v2.y);
}
// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 operator-(const sycl::double2 &v1,
// CHECK:                                              const sycl::double2 &v2) {
// CHECK:   return sycl::double2(v1.x() - v2.x(), v1.y() - v2.y());
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator-(const double2 &v1,
                                             const double2 &v2) {
  return make_double2(v1.x - v2.x, v1.y - v2.y);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator-(const sycl::double2 &v1,
// CHECK:                                              const sycl::double2 &v2) {
// CHECK:   return sycl::double2(v1.x() - v2.x(), v1.y() - v2.y());
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator-(const double2 &v1,
                                             const double2 &v2) {
  return make_double2(v1.x - v2.x, v1.y - v2.y);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 operator*(const sycl::double2 &v,
// CHECK:                                              const double &r) {
// CHECK:   return sycl::double2(v.x() * r, v.y() * r);
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator*(const double2 &v,
                                             const double &r) {
  return make_double2(v.x * r, v.y * r);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator*(const sycl::double2 &v,
// CHECK:                                              const double &r) {
// CHECK:   return sycl::double2(v.x() * r, v.y() * r);
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator*(const double2 &v,
                                             const double &r) {
  return make_double2(v.x * r, v.y * r);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 operator*(const double &r,
// CHECK:                                              const sycl::double2 &v) {
// CHECK:   return sycl::double2(v.x() * r, v.y() * r);
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator*(const double &r,
                                             const double2 &v) {
  return make_double2(v.x * r, v.y * r);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator*(const double &r,
// CHECK:                                              const sycl::double2 &v) {
// CHECK:   return sycl::double2(v.x() * r, v.y() * r);
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator*(const double &r,
                                             const double2 &v) {
  return make_double2(v.x * r, v.y * r);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: inline sycl::double2 operator/(const sycl::double2 &v,
// CHECK:                                              const double &r) {
// CHECK:   double rinv = (double)1. / r;
// CHECK:   return sycl::double2(v.x() * rinv, v.y() * rinv);
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
__host__ __device__ inline double2 operator/(const double2 &v,
                                             const double &r) {
  double rinv = (double)1. / r;
  return make_double2(v.x * rinv, v.y * rinv);
}

// CHECK:/*
// CHECK-NEXT:DPCT1011:{{[0-9]+}}: The tool detected overloaded operators for built-in vector types, which may conflict with the SYCL 2020 standard operators (see 4.14.2.1 Vec interface). The tool inserted a namespace to avoid the conflict. Use SYCL 2020 standard operators instead.
// CHECK-NEXT:*/
// CHECK: namespace dpct_operator_overloading {
// CHECK: template<typename T>
// CHECK: inline sycl::double2 operator/(const sycl::double2 &v,
// CHECK:                                              const double &r) {
// CHECK:   double rinv = (double)1. / r;
// CHECK:   return sycl::double2(v.x() * rinv, v.y() * rinv);
// CHECK: }
// CHECK: }  // namespace dpct_operator_overloading
template<typename T>
__host__ __device__ inline double2 operator/(const double2 &v,
                                             const double &r) {
  double rinv = (double)1. / r;
  return make_double2(v.x * rinv, v.y * rinv);
}

// CHECK: inline double dot(const sycl::double2 &v1, const sycl::double2 &v2) {
// CHECK:   return (v1.x() * v2.x() + v1.y() * v2.y());
// CHECK: }
__host__ __device__ inline double dot(const double2 &v1, const double2 &v2) {
  return (v1.x * v2.x + v1.y * v2.y);
}

// CHECK: inline double cross(const sycl::double2 &v1, const sycl::double2 &v2) {
// CHECK:   return (v1.x() * v2.y() - v1.y() * v2.x());
// CHECK: }
__host__ __device__ inline double cross(const double2 &v1, const double2 &v2) {
  return (v1.x * v2.y - v1.y * v2.x);
}

// CHECK: inline double length(const sycl::double2 &v) {
// CHECK:   return (sycl::sqrt(v.x() * v.x() + v.y() * v.y()));
// CHECK: }
__host__ __device__ inline double length(const double2 &v) {
  return (std::sqrt(v.x * v.x + v.y * v.y));
}

// CHECK: inline double length2(const sycl::double2 &v) {
// CHECK:   return (v.x() * v.x() + v.y() * v.y());
// CHECK: }
__host__ __device__ inline double length2(const double2 &v) {
  return (v.x * v.x + v.y * v.y);
}

// CHECK: inline sycl::double2 rotateCCW(const sycl::double2 &v) {
// CHECK:   return sycl::double2(-v.y(), v.x());
// CHECK: }
__host__ __device__ inline double2 rotateCCW(const double2 &v) {
  return make_double2(-v.y, v.x);
}

// CHECK: inline sycl::double2 rotateCW(const sycl::double2 &v) {
// CHECK:   return sycl::double2(v.y(), -v.x());
// CHECK: }
__host__ __device__ inline double2 rotateCW(const double2 &v) {
  return make_double2(v.y, -v.x);
}

// CHECK: inline sycl::double2 project(sycl::double2 &v, const sycl::double2 &u) {
// CHECK:   return dpct_operator_overloading::operator-(v , dpct_operator_overloading::operator*(dot(v, u) , u));
// CHECK: }
__host__ __device__ inline double2 project(double2 &v, const double2 &u) {
  return v - dot(v, u) * u;
}

// CHECK: void test() {
// CHECK:   sycl::double2 a;
// CHECK:   sycl::double2 b;
// CHECK:   dpct_operator_overloading::operator+=(a , b);
// CHECK:   dpct_operator_overloading::operator-(a);
// CHECK:   b = a;
// CHECK: }
void test() {
  double2 a;
  double2 b;
  a += b;
  -a;
  b = a;
}

//CHECK: template<typename T> class A{
//CHECK-NEXT: public:
//CHECK-NEXT:   A operator-(const T b) const
//CHECK-NEXT:   {
//CHECK-NEXT:     return *this;
//CHECK-NEXT:   };
//CHECK-NEXT: };
template<typename T> class A{
public:
  A operator-(const T b) const
  {
    return *this;
  };
};

//CHECK: typedef A<sycl::int2> A2;
//CHECK-NEXT: void foo(){
//CHECK-NEXT:   sycl::int2 i2;
//CHECK-NEXT:   A2 a;
//CHECK-NEXT:   a - i2;
//CHECK-NEXT: }
typedef A<int2> A2;
void foo(){
  int2 i2;
  A2 a;
  a - i2;
}

inline __device__ float2 operator+(const float2 & a, const float2 & b) {
  return {a.x + b.x, a.y + b.y};
}

// CHECK: template<typename T>
// CHECK: struct Sum {
// CHECK:   inline Sum() {}
// CHECK:   inline T operator()(const T &a, const T &b) const {
// CHECK:       return a + b;
// CHECK:   }
// CHECK: };
template<typename T>
struct Sum {
  inline __device__ Sum() {}
  inline __device__ T operator()(const T &a, const T &b) const {
      return a + b;
  }
};

// CHECK: template <typename T>
// CHECK: void bar() {
// CHECK:   T a, b, c;
// CHECK:   c = a + b;
// CHECK: }
template <typename T>
__device__ void bar() {
  T a, b, c;
  c = a + b;
}

__global__ void kernel() {
  bar<float2>();
  bar<float>();
  Sum<float2> a;
  Sum<float> b;
}
