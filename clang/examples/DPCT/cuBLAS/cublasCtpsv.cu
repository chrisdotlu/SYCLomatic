
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n,
          const hipComplex *a, hipComplex *x, int incx) {
  // Start
  hipblasCtpsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, a /*const cuComplex **/, x /*cuComplex **/,
              incx /*int*/);
  // End
}
