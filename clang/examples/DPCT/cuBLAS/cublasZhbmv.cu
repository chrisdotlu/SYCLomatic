
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n, int k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *x, int incx, const hipDoubleComplex *beta,
          hipDoubleComplex *y, int incy) {
  // Start
  hipblasZhbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, k /*int*/, alpha /*const cuDoubleComplex **/,
              a /*const cuDoubleComplex **/, lda /*int*/,
              x /*const cuDoubleComplex **/, incx /*int*/,
              beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
              incy /*int*/);
  // End
}
