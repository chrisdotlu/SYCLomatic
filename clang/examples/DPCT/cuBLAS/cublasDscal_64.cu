
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const double *alpha, double *x,
          int64_t incx) {
  // Start
  hipblasDscal_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const double **/, x /*double **/, incx /*int64_t*/);
  // End
}
