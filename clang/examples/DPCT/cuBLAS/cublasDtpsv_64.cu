
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int64_t n,
          const double *a, double *x, int64_t incx) {
  // Start
  hipblasDtpsv_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
                 n /*int64_t*/, a /*const double **/, x /*double **/,
                 incx /*int64_t*/);
  // End
}
