
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int64_t lda,
          const hipDoubleComplex *b, int64_t ldb, const double *beta,
          hipDoubleComplex *c, int64_t ldc) {
  // Start
  hipblasZher2k_64(
      handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
      trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
      alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
      lda /*int64_t*/, b /*const cuDoubleComplex **/, ldb /*int64_t*/,
      beta /*const double **/, c /*cuDoubleComplex **/, ldc /*int64_t*/);
  // End
}
